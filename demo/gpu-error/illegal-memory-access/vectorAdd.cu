#include "hip/hip_runtime.h"
/* Copyright 2017 Google Inc. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
limitations under the License.
*/

/**
 * This sample does a very simple vector add, and will trigger illegal memory
 * access error. The purpose of this sample is to test the error handling of
 * the device plugin or other components.
 */

 #include <stdio.h>
 #include <hip/hip_runtime.h>
 #include <hip/hip_runtime_api.h>

 /**
  * Computes the vector addition and intentionally triggers memory error
  */
 __global__ void
 vectorAddAndTriggerError(const float *A, const float *B, float *C, int numElements)
 {
     // Intentionally triggering out of bounds
     int i = (blockDim.x * blockIdx.x) + threadIdx.x + 1000000000000;
     C[i] = A[i] + B[i];
 }

 int main(void)
 {
     printf("Starting illegal memory access sample\n");
     // Error code to check return values for CUDA calls
     hipError_t err = hipSuccess;

     int vecLength = 50000;
     size_t size = vecLength * sizeof(float);

     // Initializing two vectors on host
     float *h_A = (float *)malloc(size);
     float *h_B = (float *)malloc(size);
     for (int i = 0; i < vecLength; ++i)
     {
         h_A[i] = rand()/(float)RAND_MAX;
         h_B[i] = rand()/(float)RAND_MAX;
     }

     // Allocating three vectors on device
     float *d_A = NULL;
     err = hipMalloc((void **)&d_A, size);
     float *d_B = NULL;
     err = hipMalloc((void **)&d_B, size);
     float *d_C = NULL;
     err = hipMalloc((void **)&d_C, size);

     // copy data from host to device
     err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
     err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

     // Run the vectorAdd func and trigger error
     int threadsPerBlock = 256;
     int blocksPerGrid =(vecLength + threadsPerBlock - 1) / threadsPerBlock;
     printf("Run vectorAdd with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
     vectorAddAndTriggerError<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, vecLength);
     err = hipGetLastError();

     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     printf("Copy results from the device to the host\n");
     float *h_C = (float *)malloc(size);
     err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

     // Expecting error here
     if (err != hipSuccess)
     {
         fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     return 0;
 }
